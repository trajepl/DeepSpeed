#include "hip/hip_runtime.h"
#include "custom_cuda_layers.h"

#define MAX_CAP 4
#define MAX_SEQ 2048

inline __device__ float relu(const float x) { return x < 0 ? 0 : x; }

__global__ void fused_bias_relu(float* input,
                                const float* bias,
                                int total_count,
                                int intermediate_size)
{
    float4* input_cast = reinterpret_cast<float4*>(input);
    const float4* bias_cast = reinterpret_cast<const float4*>(bias);
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float4 data = input_cast[offset];
        float4 bias_data = bias_cast[offset % intermediate_size];

        data.x += bias_data.x;
        data.y += bias_data.y;
        data.z += bias_data.z;
        data.w += bias_data.w;

        data.x = relu(data.x);
        data.y = relu(data.y);
        data.z = relu(data.z);
        data.w = relu(data.w);

        input_cast[offset] = data;
    }
}

__global__ void fused_bias_relu(__half* input,
                                const __half* bias,
                                int total_count,
                                int intermediate_size)
{
#ifdef HALF_PRECISION_AVAILABLE

    float2* input_cast = reinterpret_cast<float2*>(input);
    const float2* bias_cast = reinterpret_cast<const float2*>(bias);

    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float2 vals_vec = input_cast[offset];
        float2 bias_vec = bias_cast[offset % intermediate_size];

        __half2* vals_half = reinterpret_cast<__half2*>(&vals_vec);
        __half2* bias_half = reinterpret_cast<__half2*>(&bias_vec);

        float2 low_data = __half22float2(vals_half[0]);
        float2 high_data = __half22float2(vals_half[1]);

        float2 low_bias = __half22float2(bias_half[0]);
        float2 high_bias = __half22float2(bias_half[1]);

        low_data.x += low_bias.x;
        low_data.y += low_bias.y;
        high_data.x += high_bias.x;
        high_data.y += high_bias.y;

        low_data.x = relu(low_data.x);
        low_data.y = relu(low_data.y);
        high_data.x = relu(high_data.x);
        high_data.y = relu(high_data.y);

        vals_half[0] = __float22half2_rn(low_data);
        vals_half[1] = __float22half2_rn(high_data);

        input_cast[offset] = vals_vec;
    }
#endif
}

template <typename T>
void launch_bias_relu(T* input,
                      const T* bias,
                      int intermediate_size,
                      int batch_size,
                      hipStream_t stream)
{
    int total_count = batch_size * (intermediate_size / 4);
    int threads = 1024;  // intermediate_size / iterations / 4;
    dim3 block_dims(threads);
    dim3 grid_dims(((total_count - 1) / 1024 + 1));  // (batch_size);

    fused_bias_relu<<<grid_dims, block_dims, 0, stream>>>(
        input, bias, total_count, intermediate_size / 4);
}

template void launch_bias_relu<float>(float*, const float*, int, int, hipStream_t);
template void launch_bias_relu<__half>(__half*, const __half*, int, int, hipStream_t);
